#include "hip/hip_runtime.h"
#include <numeric>
#include "cuFFTAdapter.cuh"

#define numBlocks(n) (((BLOCK_SIZE) + (n) - 1) / (BLOCK_SIZE))
#define BLOCK_SIZE 1024

CuFFTAdapter::CuFFTAdapter(std::tuple<int, int, int> dims)
    : length(std::get<0>(dims) * std::get<1>(dims) * std::get<2>(dims)) {
  hipfftPlan3d(&plan, std::get<2>(dims), std::get<1>(dims), std::get<0>(dims), HIPFFT_C2C);
}

void CuFFTAdapter::free() {
  hipfftDestroy(plan);
}

void CuFFTAdapter::fft(hipComplex* d_in, hipComplex* d_out) {
  hipfftExecC2C(plan, d_in, d_out, HIPFFT_FORWARD);
}

__global__ void scaleAfterInverse(hipComplex* data, int length) {
  for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < length;
       idx += gridDim.x * blockDim.x) {
    data[idx].x /= length;
    data[idx].y /= length;
  }
}

void CuFFTAdapter::ifft(hipComplex* d_in, hipComplex* d_out) {
  hipfftExecC2C(plan, d_in, d_out, HIPFFT_BACKWARD);
  scaleAfterInverse<<<numBlocks(length), BLOCK_SIZE>>>(d_out, length);
}
