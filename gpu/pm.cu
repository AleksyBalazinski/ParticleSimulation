#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <algorithm>
#include <execution>
#include <iostream>
#include "conversions.cuh"
#include "disk_sampler_linear.cuh"
#include "external_fields.cuh"
#include "helper_macros.h"
#include "measure_time.cuh"
#include "pm.cuh"
#include "settings.cuh"
#include "state_recorder.cuh"
#include "utils.cuh"

#define CELLS_CNT ((NGx) * (NGy) * (NGz))
#define BLOCK_SIZE 1024
#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 8
#define BLOCK_SIZE_Z 8
#define NUM_BLOCKS(n) (((BLOCK_SIZE) + (n) - 1) / (BLOCK_SIZE))
#define GRID_IDX(x, y, z) ((x) + (y) * (NGx) + (z) * (NGx) * (NGy))
#define USE_SMEM

declareCudaTimer(reassignDensity);
declareCudaTimer(forwardFFT);
declareCudaTimer(findFourierPotential);
declareCudaTimer(inverseFFT);
declareCudaTimer(scaleAfterInverse);
declareCudaTimer(findFieldInCells);
declareCudaTimer(updateAccelerations);

declareHostTimer(memcpy);
declareHostTimer(recordState);
declareHostTimer(pm);
declareHostTimer(boundsCheck);

dim3 block;
dim3 grid;

__device__ inline int mod(int a, int b) {
  return (a % b + b) % b;
}

__device__ inline int modIndex(int i, int j, int k) {
  return mod(i, NGx) + mod(j, NGy) * NGx + mod(k, NGz) * NGx * NGy;
}

bool isWithinBox(const Vec3 pos, Triple<float> boxSize) {
  return pos.x >= 0 && pos.x <= boxSize.x && pos.y >= 0 && pos.y <= boxSize.y && pos.z >= 0 &&
         pos.z <= boxSize.z;
}

__global__ void reassignDensity(hipfftComplex* gridDensity,
                                Vec3* positions,
                                float* masses,
                                float H,
                                float DT,
                                float G) {
  for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < N; idx += blockDim.x * gridDim.x) {
    int particleIdx = idx;

    int x = (int)positions[particleIdx].x;
    int y = (int)positions[particleIdx].y;
    int z = (int)positions[particleIdx].z;

    float vol = H * H * H;
    float d = densityToCodeUnits(masses[particleIdx] / vol, DT, G);

    float dx = positions[particleIdx].x - x;
    float dy = positions[particleIdx].y - y;
    float dz = positions[particleIdx].z - z;
    float tx = 1 - dx;
    float ty = 1 - dy;
    float tz = 1 - dz;

    atomicAdd(reinterpret_cast<float*>(gridDensity) + 2 * GRID_IDX(x, y, z), d * tx * ty * tz);
    atomicAdd(reinterpret_cast<float*>(gridDensity) + 2 * GRID_IDX(x + 1, y, z), d * dx * ty * tz);
    atomicAdd(reinterpret_cast<float*>(gridDensity) + 2 * GRID_IDX(x, y + 1, z), d * tx * dy * tz);
    atomicAdd(reinterpret_cast<float*>(gridDensity) + 2 * GRID_IDX(x, y, z + 1), d * tx * ty * dz);

    atomicAdd(reinterpret_cast<float*>(gridDensity) + 2 * GRID_IDX(x + 1, y + 1, z),
              d * dx * dy * tz);
    atomicAdd(reinterpret_cast<float*>(gridDensity) + 2 * GRID_IDX(x + 1, y, z + 1),
              d * dx * ty * dz);
    atomicAdd(reinterpret_cast<float*>(gridDensity) + 2 * GRID_IDX(x, y + 1, z + 1),
              d * tx * dy * dz);

    atomicAdd(reinterpret_cast<float*>(gridDensity) + 2 * GRID_IDX(x + 1, y + 1, z + 1),
              d * dx * dy * dz);
  }
}

__host__ __device__ Triple<int> indexTripleFromFlat(int flatIndex) {
  int x = flatIndex % NGx;
  int y = (flatIndex / NGx) % NGy;
  int z = flatIndex / (NGx * NGy);
  return Triple(x, y, z);
}

__global__ void findFourierPotential(hipfftComplex* potentialFourier,
                                     hipfftComplex* gridDensityFourier) {
  for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < CELLS_CNT;
       idx += blockDim.x * gridDim.x) {
    auto [kx, ky, kz] = indexTripleFromFlat(idx);
    if (kx == 0 && ky == 0 && kz == 0) {
      return;
    }
    float sx = sinf(PI * kx / NGx);
    float sy = sinf(PI * ky / NGy);
    float sz = sinf(PI * kz / NGz);
    float green = -0.25f / (sx * sx + sy * sy + sz * sz);

    hipfftComplex densityFourier = gridDensityFourier[GRID_IDX(kx, ky, kz)];
    potentialFourier[GRID_IDX(kx, ky, kz)] =
        make_hipComplex(green * densityFourier.x, green * densityFourier.y);
  }
}

__global__ void scaleAfterInverse(hipfftComplex* gridPotential) {
  for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < CELLS_CNT;
       idx += gridDim.x * blockDim.x) {
    gridPotential[idx].x /= CELLS_CNT;
  }
}

__global__ void findFieldInCells(Vec3* gridField, hipfftComplex* gridPotential) {
#ifdef USE_SMEM
  __shared__ float smem_potential[BLOCK_SIZE_X + 2][BLOCK_SIZE_Y + 2][BLOCK_SIZE_Z + 2];
#endif
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int idy = threadIdx.y + blockDim.y * blockIdx.y;
  int idz = threadIdx.z + blockDim.z * blockIdx.z;

  if ((idx < NGx) && (idy < NGy) && (idz < NGz)) {
#ifdef USE_SMEM
    int x = threadIdx.x + 1;
    int y = threadIdx.y + 1;
    int z = threadIdx.z + 1;

    smem_potential[x][y][z] = gridPotential[GRID_IDX(idx, idy, idz)].x;

    __syncthreads();

    float xl, yl, zl;
    float xr, yr, zr;

    xl = (x == 1) ? gridPotential[modIndex(idx - 1, idy, idz)].x : smem_potential[x - 1][y][z];
    yl = (y == 1) ? gridPotential[modIndex(idx, idy - 1, idz)].x : smem_potential[x][y - 1][z];
    zl = (z == 1) ? gridPotential[modIndex(idx, idy, idz - 1)].x : smem_potential[x][y][z - 1];
    xr = (x == BLOCK_SIZE_X) ? gridPotential[modIndex(idx + 1, idy, idz)].x
                             : smem_potential[x + 1][y][z];
    yr = (y == BLOCK_SIZE_Y) ? gridPotential[modIndex(idx, idy + 1, idz)].x
                             : smem_potential[x][y + 1][z];
    zr = (z == BLOCK_SIZE_Z) ? gridPotential[modIndex(idx, idy, idz + 1)].x
                             : smem_potential[x][y][z + 1];

    gridField[GRID_IDX(idx, idy, idz)].x = -0.5f * (xr - xl);
    gridField[GRID_IDX(idx, idy, idz)].y = -0.5f * (yr - yl);
    gridField[GRID_IDX(idx, idy, idz)].z = -0.5f * (zr - zl);
#else
    float xl, yl, zl;
    float xr, yr, zr;

    xl = (idx == 0) ? gridPotential[modIndex(idx - 1, idy, idz)].x
                    : gridPotential[GRID_IDX(idx - 1, idy, idz)].x;
    yl = (idy == 0) ? gridPotential[modIndex(idx, idy - 1, idz)].x
                    : gridPotential[GRID_IDX(idx, idy - 1, idz)].x;
    zl = (idz == 0) ? gridPotential[modIndex(idx, idy, idz - 1)].x
                    : gridPotential[GRID_IDX(idx, idy, idz - 1)].x;
    xr = (idx == NG - 1) ? gridPotential[modIndex(idx + 1, idy, idz)].x
                         : gridPotential[GRID_IDX(idx + 1, idy, idz)].x;
    yr = (idy == NG - 1) ? gridPotential[modIndex(idx, idy + 1, idz)].x
                         : gridPotential[GRID_IDX(idx, idy + 1, idz)].x;
    zr = (idz == NG - 1) ? gridPotential[modIndex(idx, idy, idz + 1)].x
                         : gridPotential[GRID_IDX(idx, idy, idz + 1)].x;

    gridField[GRID_IDX(idx, idy, idz)].x = -0.5f * (xr - xl);
    gridField[GRID_IDX(idx, idy, idz)].y = -0.5f * (yr - yl);
    gridField[GRID_IDX(idx, idy, idz)].z = -0.5f * (zr - zl);
#endif
  }
}

__device__ Vec3 interpolateField(Vec3* gridField, Vec3 position) {
  int xi = (int)position.x;
  int yi = (int)position.y;
  int zi = (int)position.z;
  float dx = position.x - xi;
  float dy = position.y - yi;
  float dz = position.z - zi;
  float tx = 1 - dx;
  float ty = 1 - dy;
  float tz = 1 - dz;

  auto field000 = gridField[GRID_IDX(xi, yi, zi)];
  auto field100 = gridField[GRID_IDX(xi + 1, yi, zi)];
  auto field010 = gridField[GRID_IDX(xi, yi + 1, zi)];
  auto field001 = gridField[GRID_IDX(xi, yi, zi + 1)];
  auto field110 = gridField[GRID_IDX(xi + 1, yi + 1, zi)];
  auto field101 = gridField[GRID_IDX(xi + 1, yi, zi + 1)];
  auto field011 = gridField[GRID_IDX(xi, yi + 1, zi + 1)];
  auto field111 = gridField[GRID_IDX(xi + 1, yi + 1, zi + 1)];

  float resX = tx * ty * tz * field000.x + dx * ty * tz * field100.x + tx * dy * tz * field010.x +
               tx * ty * dz * field001.x + dx * dy * tz * field110.x + dx * ty * dz * field101.x +
               tx * dy * dz * field011.x + dx * dy * dz * field111.x;

  float resY = tx * ty * tz * field000.y + dx * ty * tz * field100.y + tx * dy * tz * field010.y +
               tx * ty * dz * field001.y + dx * dy * tz * field110.y + dx * ty * dz * field101.y +
               tx * dy * dz * field011.y + dx * dy * dz * field111.y;

  float resZ = tx * ty * tz * field000.z + dx * ty * tz * field100.z + tx * dy * tz * field010.z +
               tx * ty * dz * field001.z + dx * dy * tz * field110.z + dx * ty * dz * field101.z +
               tx * dy * dz * field011.z + dx * dy * dz * field111.z;

  return Vec3(resX, resY, resZ);
}

__global__ void updateAccelerations(Vec3* accelerations,
                                    Vec3* positions,
                                    Vec3* gridField,
                                    SphRadDecrFieldParams bulgeParams,
                                    float G,
                                    float H,
                                    float DT) {
  for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < N; idx += blockDim.x * gridDim.x) {
    Vec3 intField = interpolateField(gridField, positions[idx]);
    Vec3 extField = accelerationToCodeUnits(
        sphRadDecrField(positionToOrigUnits(positions[idx], H), bulgeParams, G), H, DT);

    accelerations[idx].x = intField.x + extField.x;
    accelerations[idx].y = intField.y + extField.y;
    accelerations[idx].z = intField.z + extField.z;
  }
}

__global__ void updateVelocities(Vec3* velocities, Vec3* accelerations) {
  for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < N; idx += blockDim.x * gridDim.x) {
    velocities[idx].x += accelerations[idx].x;
    velocities[idx].y += accelerations[idx].y;
    velocities[idx].z += accelerations[idx].z;
  }
}

void pmMethodStep(Vec3* d_accelerations,
                  hipfftComplex* d_gridDensity,
                  hipfftComplex* d_gridDensityFourier,
                  hipfftComplex* d_gridPotential,
                  hipfftComplex* d_gridPotentialFourier,
                  Vec3* d_gridField,
                  Vec3* d_positions,
                  float* d_masses,
                  SphRadDecrFieldParams bulgeParams,
                  float H,
                  float DT,
                  float G) {
  hipMemset(d_gridDensity, 0, CELLS_CNT * sizeof(hipfftComplex));
  measureCudaTime(reassignDensity, reassignDensity<<<NUM_BLOCKS(N), BLOCK_SIZE>>>(
                                       d_gridDensity, d_positions, d_masses, H, DT, G));

  hipfftHandle plan = 0;
  hipfftPlan3d(&plan, NGz, NGy, NGx, HIPFFT_C2C);
  measureCudaTime(forwardFFT,
                  hipfftExecC2C(plan, d_gridDensity, d_gridDensityFourier, HIPFFT_FORWARD));

  hipMemset(d_gridPotentialFourier, 0, sizeof(hipfftComplex) * CELLS_CNT);
  measureCudaTime(findFourierPotential, findFourierPotential<<<NUM_BLOCKS(CELLS_CNT), BLOCK_SIZE>>>(
                                            d_gridPotentialFourier, d_gridDensityFourier));

  plan = 0;
  hipfftPlan3d(&plan, NGz, NGy, NGx, HIPFFT_C2C);
  measureCudaTime(inverseFFT,
                  hipfftExecC2C(plan, d_gridPotentialFourier, d_gridPotential, HIPFFT_BACKWARD));
  scaleAfterInverse<<<NUM_BLOCKS(CELLS_CNT), BLOCK_SIZE>>>(d_gridPotential);

  measureCudaTime(findFieldInCells,
                  findFieldInCells<<<grid, block>>>(d_gridField, d_gridPotential));

  measureCudaTime(updateAccelerations,
                  updateAccelerations<<<NUM_BLOCKS(N), BLOCK_SIZE>>>(
                      d_accelerations, d_positions, d_gridField, bulgeParams, G, H, DT));

  accCudaTime(reassignDensity);
  accCudaTime(forwardFFT);
  accCudaTime(findFourierPotential);
  accCudaTime(inverseFFT);
  accCudaTime(findFieldInCells);
  accCudaTime(updateAccelerations);
}

__global__ void setHalfVelocities(Vec3* velocities, Vec3* accelerations) {
  for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < N; idx += blockDim.x * gridDim.x) {
    velocities[idx] += 0.5f * accelerations[idx];
  }
}

__global__ void updatePositions(Vec3* positions, Vec3* velocities) {
  for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < N; idx += blockDim.x * gridDim.x) {
    positions[idx] += velocities[idx];
  }
}

inline float toMs(const std::chrono::nanoseconds delta) {
  return delta.count() * 1e-6f;
}

void pmMethod(std::vector<Vec3>& state,
              const std::vector<float>& masses,
              Triple<float> effectiveBoxSize,
              float H,
              float DT,
              SphRadDecrFieldParams bulgeParams,
              float G,
              int simLength) {
  block.x = BLOCK_SIZE_X;
  block.y = BLOCK_SIZE_Y;
  block.z = BLOCK_SIZE_Z;
  grid.x = (64 + block.x - 1) / block.x;
  grid.y = (64 + block.y - 1) / block.y;
  grid.z = (64 + block.z - 1) / block.z;

  createCudaEvents(reassignDensity);
  createCudaEvents(forwardFFT);
  createCudaEvents(findFourierPotential);
  createCudaEvents(inverseFFT);
  createCudaEvents(scaleAfterInverse);
  createCudaEvents(findFieldInCells);
  createCudaEvents(updateAccelerations);

  StateRecorder stateRecorder("output_gpu.txt", "a", "b");

  float* d_masses;
  Vec3* d_positions;
  Vec3* d_velocities;
  Vec3* d_accelerations;

  hipfftComplex* d_gridDensity;
  hipfftComplex* d_gridDensityFourier;
  hipfftComplex* d_gridPotential;
  hipfftComplex* d_gridPotentialFourier;
  Vec3* d_gridField;

  hipMalloc(&d_masses, N * sizeof(float));
  hipMalloc(&d_positions, N * sizeof(Vec3));
  hipMalloc(&d_velocities, N * sizeof(Vec3));
  hipMalloc(&d_accelerations, N * sizeof(Vec3));

  hipMalloc(&d_gridDensity, CELLS_CNT * sizeof(hipfftComplex));
  hipMalloc(&d_gridDensityFourier, CELLS_CNT * sizeof(hipfftComplex));
  hipMalloc(&d_gridPotential, CELLS_CNT * sizeof(hipfftComplex));
  hipMalloc(&d_gridPotentialFourier, CELLS_CNT * sizeof(hipfftComplex));
  hipMalloc(&d_gridField, CELLS_CNT * sizeof(Vec3));

  hipMemcpy(d_positions, state.data(), N * sizeof(Vec3), hipMemcpyHostToDevice);
  hipMemcpy(d_velocities, state.data() + N, N * sizeof(Vec3), hipMemcpyHostToDevice);
  hipMemcpy(d_masses, masses.data(), N * sizeof(float), hipMemcpyHostToDevice);

  stateToCodeUnits<<<NUM_BLOCKS(N), BLOCK_SIZE>>>(d_positions, d_velocities, H, DT, N);

  pmMethodStep(d_accelerations, d_gridDensity, d_gridDensityFourier, d_gridPotential,
               d_gridPotentialFourier, d_gridField, d_positions, d_masses, bulgeParams, H, DT, G);

  setHalfVelocities<<<NUM_BLOCKS(N), BLOCK_SIZE>>>(d_velocities, d_accelerations);

  for (int t = 0; t <= simLength; ++t) {
    hipDeviceSynchronize();
    std::cout << "progress: " << float(t) / simLength << '\r';
    std::cout.flush();

    updatePositions<<<NUM_BLOCKS(N), BLOCK_SIZE>>>(d_positions, d_velocities);

    stateToOrigUnits<<<NUM_BLOCKS(N), BLOCK_SIZE>>>(d_positions, d_velocities, H, DT, N);

    measureHostTime(
        memcpy, hipMemcpy(state.data(), d_positions, N * sizeof(Vec3), hipMemcpyDeviceToHost));

    measureHostTime(recordState, stateRecorder.recordPositions(state.begin(), state.begin() + N));

    bool movedOutside = false;
    measureHostTime(boundsCheck, movedOutside = std::any_of(
                                     std::execution::par_unseq, state.begin(), state.begin() + N,
                                     [effectiveBoxSize](const Vec3& pos) {
                                       return !isWithinBox(pos, effectiveBoxSize);
                                     }));
    if (movedOutside) {
      std::cout << "Particle moved outside the grid.\n";
      break;
    }

    stateToCodeUnits<<<NUM_BLOCKS(N), BLOCK_SIZE>>>(d_positions, d_velocities, H, DT, N);

    measureHostTime(pm, pmMethodStep(d_accelerations, d_gridDensity, d_gridDensityFourier,
                                     d_gridPotential, d_gridPotentialFourier, d_gridField,
                                     d_positions, d_masses, bulgeParams, H, DT, G));
    updateVelocities<<<NUM_BLOCKS(N), BLOCK_SIZE>>>(d_velocities, d_accelerations);
  }
  hipDeviceSynchronize();
  stateRecorder.flush();

  printCudaTime(reassignDensity);
  printCudaTime(forwardFFT);
  printCudaTime(findFourierPotential);
  printCudaTime(inverseFFT);
  printCudaTime(findFieldInCells);
  printCudaTime(updateAccelerations);
  printHostTime(pm);
  printHostTime(memcpy);
  printHostTime(boundsCheck);
  printHostTime(recordState);

  hipFree(d_masses);
  hipFree(d_positions);
  hipFree(d_velocities);
  hipFree(d_accelerations);

  hipFree(d_gridDensity);
  hipFree(d_gridDensityFourier);
  hipFree(d_gridPotential);
  hipFree(d_gridPotentialFourier);
  hipFree(d_gridField);

  destroyCudaEvents(reassignDensity);
  destroyCudaEvents(forwardFFT);
  destroyCudaEvents(findFourierPotential);
  destroyCudaEvents(inverseFFT);
  destroyCudaEvents(scaleAfterInverse);
  destroyCudaEvents(findFieldInCells);
  destroyCudaEvents(updateAccelerations);
}