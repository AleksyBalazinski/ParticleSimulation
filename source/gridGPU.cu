#include "hip/hip_runtime.h"
#include "gridGPU.cuh"

GridGPU::GridGPU(std::tuple<int, int, int> gridPoints)
    : gridPointsX(std::get<0>(gridPoints)),
      gridPointsY(std::get<1>(gridPoints)),
      gridPointsZ(std::get<2>(gridPoints)),
      length(gridPointsX * gridPointsY * gridPointsZ),
      fftAdapter(gridPoints) {
  hipMalloc(&d_field, length * sizeof(Vec3));
  hipMalloc(&d_density, length * sizeof(hipfftComplex));
  hipMalloc(&d_densityFourier, length * sizeof(hipfftComplex));
  hipMalloc(&d_potential, length * sizeof(hipfftComplex));
  hipMalloc(&d_potentialFourier, length * sizeof(hipfftComplex));
  hipMalloc(&d_greensFunction, length * sizeof(hipfftComplex));
}

void GridGPU::freeGrid() {
  fftAdapter.free();

  hipFree(d_field);
  hipFree(d_density);
  hipFree(d_densityFourier);
  hipFree(d_potential);
  hipFree(d_potentialFourier);
  hipFree(d_greensFunction);
}

__device__ void GridGPU::assignDensity(int x, int y, int z, float density) {
  atomicAdd((float*)(d_density) + 2 * getIndx(x, y, z), density);
}

void GridGPU::clearDensity() {
  hipMemset(d_density, 0, length * sizeof(hipfftComplex));
}
