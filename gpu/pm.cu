#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <algorithm>
#include <chrono>
#include <execution>
#include <iostream>
#include "common.h"
#include "conversions.cuh"
#include "disk_sampler_linear.cuh"
#include "helper_macros.h"
#include "state_recorder.cuh"
#include "utils.cuh"

#define CELLS_CNT ((NG) * (NG) * (NG))
#define NUM_BLOCKS(n) (((BLOCK_SIZE) + (n) - 1) / (BLOCK_SIZE))
#define GRID_IDX(x, y, z) ((x) + (y) * (NG) + (z) * (NG) * (NG))
#define GROUP_SIZE 2
#define USE_SMEM

__device__ int mod(int a, int b) {
  return (a % b + b) % b;
}

__device__ int modIndex(int i, int j, int k) {
  return mod(i, NG) + mod(j, NG) * NG + mod(k, NG) * NG * NG;
}

bool isWithinBox(const Vec3 pos, float boxSize) {
  return pos.x >= 0 && pos.x <= boxSize && pos.y >= 0 && pos.y <= boxSize && pos.z >= 0 &&
         pos.z <= boxSize;
}

__global__ void reassignDensity(hipfftComplex* gridDensity,
                                Vec3* positions,
                                float* masses,
                                float H,
                                float DT,
                                float G) {
  for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < N; idx += blockDim.x * gridDim.x) {
    int particleIdx = idx;

    int x = (int)positions[particleIdx].x;
    int y = (int)positions[particleIdx].y;
    int z = (int)positions[particleIdx].z;

    float vol = H * H * H;
    float d = densityToCodeUnits(masses[particleIdx] / vol, DT, G);

    float dx = positions[particleIdx].x - x;
    float dy = positions[particleIdx].y - y;
    float dz = positions[particleIdx].z - z;
    float tx = 1 - dx;
    float ty = 1 - dy;
    float tz = 1 - dz;

    atomicAdd(reinterpret_cast<float*>(gridDensity) + 2 * GRID_IDX(x, y, z), d * tx * ty * tz);
    atomicAdd(reinterpret_cast<float*>(gridDensity) + 2 * GRID_IDX(x + 1, y, z), d * dx * ty * tz);
    atomicAdd(reinterpret_cast<float*>(gridDensity) + 2 * GRID_IDX(x, y + 1, z), d * tx * dy * tz);
    atomicAdd(reinterpret_cast<float*>(gridDensity) + 2 * GRID_IDX(x, y, z + 1), d * tx * ty * dz);

    atomicAdd(reinterpret_cast<float*>(gridDensity) + 2 * GRID_IDX(x + 1, y + 1, z),
              d * dx * dy * tz);
    atomicAdd(reinterpret_cast<float*>(gridDensity) + 2 * GRID_IDX(x + 1, y, z + 1),
              d * dx * ty * dz);
    atomicAdd(reinterpret_cast<float*>(gridDensity) + 2 * GRID_IDX(x, y + 1, z + 1),
              d * tx * dy * dz);

    atomicAdd(reinterpret_cast<float*>(gridDensity) + 2 * GRID_IDX(x + 1, y + 1, z + 1),
              d * dx * dy * dz);
  }
}

__global__ void findFourierPotential(hipfftComplex* potentialFourier,
                                     hipfftComplex* gridDensityFourier) {
  for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < CELLS_CNT;
       idx += blockDim.x * gridDim.x) {
    int kx = idx % NG;
    int ky = (idx / NG) % NG;
    int kz = idx / (NG * NG);
    if (kx == 0 && ky == 0 && kz == 0) {
      return;
    }
    float sx = sinf(PI * kx / NG);
    float sy = sinf(PI * ky / NG);
    float sz = sinf(PI * kz / NG);
    float green = -0.25f / (sx * sx + sy * sy + sz * sz);

    hipfftComplex densityFourier = gridDensityFourier[GRID_IDX(kx, ky, kz)];
    potentialFourier[GRID_IDX(kx, ky, kz)] =
        make_hipComplex(green * densityFourier.x, green * densityFourier.y);
  }
}

__device__ Vec3 getFieldInCell(int x, int y, int z, hipfftComplex* gridPotential) {
  Vec3 field;
  field.x =
      -0.5f * (gridPotential[modIndex(x + 1, y, z)].x - gridPotential[modIndex(x - 1, y, z)].x);
  field.y =
      -0.5f * (gridPotential[modIndex(x, y + 1, z)].x - gridPotential[modIndex(x, y - 1, z)].x);
  field.z =
      -0.5f * (gridPotential[modIndex(x, y, z + 1)].x - gridPotential[modIndex(x, y, z - 1)].x);

  return field;
}

__global__ void scaleAfterInverse(hipfftComplex* gridPotential) {
  for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < CELLS_CNT;
       idx += gridDim.x * blockDim.x) {
    gridPotential[idx].x /= CELLS_CNT;
  }
}

const int blockSizeX = 16;
const int blockSizeY = 8;
const int blockSizeZ = 8;

__global__ void findFieldInCells(Vec3* gridField, hipfftComplex* gridPotential) {
#ifdef USE_SMEM
  const int blockSizeX = 16;
  const int blockSizeY = 8;
  const int blockSizeZ = 8;

  __shared__ float smem_potential[blockSizeX + 2][blockSizeY + 2][blockSizeZ + 2];
  int x = threadIdx.x + 1;
  int y = threadIdx.y + 1;
  int z = threadIdx.z + 1;
#endif
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int idy = threadIdx.y + blockDim.y * blockIdx.y;
  int idz = threadIdx.z + blockDim.z * blockIdx.z;

  if ((idx < NG) && (idy < NG) && (idz < NG)) {
#ifdef USE_SMEM
    smem_potential[x][y][z] = gridPotential[GRID_IDX(idx, idy, idz)].x;
    if (threadIdx.x == 0) {
      smem_potential[0][y][z] = gridPotential[modIndex(idx - 1, idy, idz)].x;
      smem_potential[blockSizeX + 1][y][z] = gridPotential[modIndex(idx + blockSizeX, idy, idz)].x;
    }
    if (threadIdx.y == 0) {
      smem_potential[x][0][z] = gridPotential[modIndex(idx, idy - 1, idz)].x;
      smem_potential[x][blockSizeY + 1][z] = gridPotential[modIndex(idx, idy + blockSizeY, idz)].x;
    }
    if (threadIdx.z == 0) {
      smem_potential[x][y][0] = gridPotential[modIndex(idx, idy, idz - 1)].x;
      smem_potential[x][y][blockSizeZ + 1] = gridPotential[modIndex(idx, idy, idz + blockSizeZ)].x;
    }
    if (threadIdx.x == 0 && threadIdx.y == 0) {
      smem_potential[0][0][z] = gridPotential[modIndex(idx - 1, idy - 1, idz)].x;
      smem_potential[0][blockSizeY + 1][z] =
          gridPotential[modIndex(idx - 1, idy + blockSizeY, idz)].x;
      smem_potential[blockSizeX + 1][0][z] =
          gridPotential[modIndex(idx + blockSizeX, idy - 1, idz)].x;
      smem_potential[blockSizeX + 1][blockSizeY + 1][z] =
          gridPotential[modIndex(idx + blockSizeX, idy + blockSizeY, idz)].x;
    }
    if (threadIdx.x == 0 && threadIdx.z == 0) {
      smem_potential[0][y][0] = gridPotential[modIndex(idx - 1, idy, idz - 1)].x;
      smem_potential[0][y][blockSizeZ + 1] =
          gridPotential[modIndex(idx - 1, idy, idz + blockSizeZ)].x;
      smem_potential[blockSizeX + 1][y][0] =
          gridPotential[modIndex(idx + blockSizeX, idy, idz - 1)].x;
      smem_potential[blockSizeX + 1][y][blockSizeZ + 1] =
          gridPotential[modIndex(idx + blockSizeX, idy, idz + blockSizeZ)].x;
    }
    if (threadIdx.y == 0 && threadIdx.z == 0) {
      smem_potential[x][0][0] = gridPotential[modIndex(idx, idy - 1, idz - 1)].x;
      smem_potential[x][0][blockSizeZ + 1] =
          gridPotential[modIndex(idx, idy - 1, idz + blockSizeZ)].x;
      smem_potential[x][blockSizeY + 1][0] =
          gridPotential[modIndex(idx, idy + blockSizeY, idz - 1)].x;
      smem_potential[x][blockSizeY + 1][blockSizeZ + 1] =
          gridPotential[modIndex(idx, idy + blockSizeY, idz + blockSizeZ)].x;
    }
    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
      smem_potential[0][0][0] = gridPotential[modIndex(idx - 1, idy - 1, idz - 1)].x;
      smem_potential[0][0][blockSizeZ + 1] =
          gridPotential[modIndex(idx - 1, idy - 1, idz + blockSizeZ)].x;
      smem_potential[0][blockSizeY + 1][0] =
          gridPotential[modIndex(idx - 1, idy + blockSizeY, idz - 1)].x;
      smem_potential[blockSizeX + 1][0][0] =
          gridPotential[modIndex(idx + blockSizeX, idy - 1, idz - 1)].x;
      smem_potential[0][blockSizeY + 1][blockSizeZ + 1] =
          gridPotential[modIndex(idx - 1, idy + blockSizeY, idz + blockSizeZ)].x;
      smem_potential[blockSizeX + 1][0][blockSizeZ + 1] =
          gridPotential[modIndex(idx + blockSizeX, idy - 1, idz + blockSizeZ)].x;
      smem_potential[blockSizeX + 1][blockSizeY + 1][0] =
          gridPotential[modIndex(idx + blockSizeX, idy + blockSizeY, idz - 1)].x;
      smem_potential[blockSizeX + 1][blockSizeY + 1][blockSizeZ + 1] =
          gridPotential[modIndex(idx + blockSizeX, idy + blockSizeY, idz + blockSizeZ)].x;
    }

    __syncthreads();

    gridField[GRID_IDX(idx, idy, idz)].x =
        -0.5f * (smem_potential[x + 1][y][z] - smem_potential[x - 1][y][z]);
    gridField[GRID_IDX(idx, idy, idz)].y =
        -0.5f * (smem_potential[x][y + 1][z] - smem_potential[x][y - 1][z]);
    gridField[GRID_IDX(idx, idy, idz)].z =
        -0.5f * (smem_potential[x][y][z + 1] - smem_potential[x][y][z - 1]);
#else
    gridField[GRID_IDX(idx, idy, idz)] = getFieldInCell(idx, idy, idz, gridPotential);
#endif
  }
}

__global__ void findFieldInCells2(Vec3* gridField, hipfftComplex* gridPotential) {
  for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < CELLS_CNT;
       idx += gridDim.x * blockDim.x) {
    int x = idx % NG;
    int y = (idx / NG) % NG;
    int z = idx / (NG * NG);

    gridField[idx] = getFieldInCell(x, y, z, gridPotential);
  }
}

__device__ Vec3 interpolateField(Vec3* gridField, Vec3 position) {
  int xi = (int)position.x;
  int yi = (int)position.y;
  int zi = (int)position.z;
  float dx = position.x - xi;
  float dy = position.y - yi;
  float dz = position.z - zi;
  float tx = 1 - dx;
  float ty = 1 - dy;
  float tz = 1 - dz;

  return tx * ty * tz * gridField[GRID_IDX(xi, yi, zi)] +
         dx * ty * tz * gridField[GRID_IDX(xi + 1, yi, zi)] +
         tx * dy * tz * gridField[GRID_IDX(xi, yi + 1, zi)] +
         tx * ty * dz * gridField[GRID_IDX(xi, yi, zi + 1)] +
         dx * dy * tz * gridField[GRID_IDX(xi + 1, yi + 1, zi)] +
         dx * ty * dz * gridField[GRID_IDX(xi + 1, yi, zi + 1)] +
         tx * dy * dz * gridField[GRID_IDX(xi, yi + 1, zi + 1)] +
         dx * dy * dz * gridField[GRID_IDX(xi + 1, yi + 1, zi + 1)];
}

__global__ void updateAccelerations(Vec3* accelerations,
                                    Vec3* positions,
                                    Vec3* gridField,
                                    float H,
                                    float DT) {
  for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < N; idx += blockDim.x * gridDim.x) {
    Vec3 intField = interpolateField(gridField, positions[idx]);
    Vec3 galaxyCenter(30, 30, 30);  // copy-paste from main
    float rb = 3.0f;
    float mb = 15.0f;
    float G = 4.5e-3f;
    Vec3 extField = accelerationToCodeUnits(
        externalFieldBulge(positionToOrigUnits(positions[idx], H), galaxyCenter, rb, mb, G), H, DT);

    accelerations[idx] = intField + extField;
  }
}

__global__ void updateVelocities(Vec3* velocities, Vec3* accelerations) {
  for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < N; idx += blockDim.x * gridDim.x) {
    velocities[idx] += accelerations[idx];
  }
}

float reassignDensityMs = 0;
float forwardFFTMs = 0;
float findFourierPotentialMs = 0;
float inverseFFTMs = 0;
float scaleAfterInverseMs = 0;
float findFieldInCellsMs = 0;
float updateAccelerationsMs = 0;

hipEvent_t reassignDensityStart, reassignDensityStop;
hipEvent_t forwardFFTStart, forwardFFTStop;
hipEvent_t findFourierPotentialStart, findFourierPotentialStop;
hipEvent_t inverseFFTStart, inverseFFTStop;
hipEvent_t scaleAfterInverseStart, scaleAfterInverseStop;
hipEvent_t findFieldInCellsStart, findFieldInCellsStop;
hipEvent_t updateAccelerationsStart, updateAccelerationsStop;

dim3 block;
dim3 grid;

void pmMethodStep(Vec3* d_accelerations,
                  hipfftComplex* d_gridDensity,
                  hipfftComplex* d_gridDensityFourier,
                  hipfftComplex* d_gridPotential,
                  hipfftComplex* d_gridPotentialFourier,
                  Vec3* d_gridField,
                  Vec3* d_positions,
                  float* d_masses,
                  float H,
                  float DT,
                  float G) {
  hipMemset(d_gridDensity, 0, CELLS_CNT * sizeof(hipfftComplex));
  cudaTime(reassignDensity, reassignDensity<<<NUM_BLOCKS(N), BLOCK_SIZE>>>(
                                d_gridDensity, d_positions, d_masses, H, DT, G));

  hipfftHandle plan = 0;
  hipfftPlan3d(&plan, NG, NG, NG, HIPFFT_C2C);
  cudaTime(forwardFFT, hipfftExecC2C(plan, d_gridDensity, d_gridDensityFourier, HIPFFT_FORWARD));

  hipMemset(d_gridPotentialFourier, 0, sizeof(hipfftComplex) * CELLS_CNT);
  cudaTime(findFourierPotential, findFourierPotential<<<NUM_BLOCKS(CELLS_CNT), BLOCK_SIZE>>>(
                                     d_gridPotentialFourier, d_gridDensityFourier));

  plan = 0;
  hipfftPlan3d(&plan, NG, NG, NG, HIPFFT_C2C);
  cudaTime(inverseFFT, hipfftExecC2C(plan, d_gridPotentialFourier, d_gridPotential, HIPFFT_BACKWARD));
  scaleAfterInverse<<<NUM_BLOCKS(CELLS_CNT), BLOCK_SIZE>>>(d_gridPotential);

  cudaTime(findFieldInCells, findFieldInCells<<<grid, block>>>(d_gridField, d_gridPotential));
  // findFieldInCells2<<<dim3(NUM_BLOCKS(CELLS_CNT)), dim3(BLOCK_SIZE)>>>(d_gridField,
  //                                                                      d_gridPotential);

  cudaTime(updateAccelerations, updateAccelerations<<<NUM_BLOCKS(N), BLOCK_SIZE>>>(
                                    d_accelerations, d_positions, d_gridField, H, DT));

  hipEventSynchronize(reassignDensityStop);
  hipEventSynchronize(forwardFFTStop);
  hipEventSynchronize(findFourierPotentialStop);
  hipEventSynchronize(inverseFFTStop);
  hipEventSynchronize(findFieldInCellsStop);
  hipEventSynchronize(updateAccelerationsStop);

  float milliseconds = 0;
  cudaAccTime(milliseconds, reassignDensity);
  cudaAccTime(milliseconds, forwardFFT);
  cudaAccTime(milliseconds, findFourierPotential);
  cudaAccTime(milliseconds, inverseFFT);
  cudaAccTime(milliseconds, findFieldInCells);
  cudaAccTime(milliseconds, updateAccelerations);
}

__global__ void setHalfVelocities(Vec3* velocities, Vec3* accelerations) {
  for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < N; idx += blockDim.x * gridDim.x) {
    velocities[idx] += 0.5f * accelerations[idx];
  }
}

__global__ void updatePositions(Vec3* positions, Vec3* velocities) {
  for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < N; idx += blockDim.x * gridDim.x) {
    positions[idx] += velocities[idx];
  }
}

long long pmTimeMs = 0;
long long memcpyTimeMs = 0;
long long boundsCheckTimeMs = 0;
long long recordStateTimeMs = 0;

long long toMs(const std::chrono::nanoseconds delta) {
  return std::chrono::duration_cast<std::chrono::milliseconds>(delta).count();
}

void pmMethod(std::vector<Vec3>& state,
              const std::vector<float>& masses,
              float effectiveBoxSize,
              float H,
              float DT,
              float G,
              int simLength) {
  block.x = blockSizeX;
  block.y = blockSizeY;
  block.z = blockSizeZ;
  grid.x = (64 + block.x - 1) / block.x;
  grid.y = (64 + block.y - 1) / block.y;
  grid.z = (64 + block.z - 1) / block.z;

  hipEventCreate(&reassignDensityStart);
  hipEventCreate(&reassignDensityStop);

  hipEventCreate(&forwardFFTStart);
  hipEventCreate(&forwardFFTStop);

  hipEventCreate(&findFourierPotentialStart);
  hipEventCreate(&findFourierPotentialStop);

  hipEventCreate(&inverseFFTStart);
  hipEventCreate(&inverseFFTStop);

  hipEventCreate(&scaleAfterInverseStart);
  hipEventCreate(&scaleAfterInverseStop);

  hipEventCreate(&findFieldInCellsStart);
  hipEventCreate(&findFieldInCellsStop);

  hipEventCreate(&updateAccelerationsStart);
  hipEventCreate(&updateAccelerationsStop);

  StateRecorder stateRecorder("output_gpu.txt", "a", "b");

  float* d_masses;
  Vec3* d_positions;
  Vec3* d_velocities;
  Vec3* d_accelerations;

  hipfftComplex* d_gridDensity;
  hipfftComplex* d_gridDensityFourier;
  hipfftComplex* d_gridPotential;
  hipfftComplex* d_gridPotentialFourier;
  Vec3* d_gridField;

  hipMalloc(&d_masses, N * sizeof(float));
  hipMalloc(&d_positions, N * sizeof(Vec3));
  hipMalloc(&d_velocities, N * sizeof(Vec3));
  hipMalloc(&d_accelerations, N * sizeof(Vec3));

  hipMalloc(&d_gridDensity, CELLS_CNT * sizeof(hipfftComplex));
  hipMalloc(&d_gridDensityFourier, CELLS_CNT * sizeof(hipfftComplex));
  hipMalloc(&d_gridPotential, CELLS_CNT * sizeof(hipfftComplex));
  hipMalloc(&d_gridPotentialFourier, CELLS_CNT * sizeof(hipfftComplex));
  hipMalloc(&d_gridField, CELLS_CNT * sizeof(Vec3));

  hipMemcpy(d_positions, state.data(), N * sizeof(Vec3), hipMemcpyHostToDevice);
  hipMemcpy(d_velocities, state.data() + N, N * sizeof(Vec3), hipMemcpyHostToDevice);
  hipMemcpy(d_masses, masses.data(), N * sizeof(float), hipMemcpyHostToDevice);

  stateToCodeUnits<<<NUM_BLOCKS(N), BLOCK_SIZE>>>(d_positions, d_velocities, H, DT, N);

  pmMethodStep(d_accelerations, d_gridDensity, d_gridDensityFourier, d_gridPotential,
               d_gridPotentialFourier, d_gridField, d_positions, d_masses, H, DT, G);

  setHalfVelocities<<<NUM_BLOCKS(N), BLOCK_SIZE>>>(d_velocities, d_accelerations);

  for (int t = 0; t <= simLength; ++t) {
    hipDeviceSynchronize();
    std::cout << "progress: " << float(t) / simLength << '\r';
    std::cout.flush();

    updatePositions<<<NUM_BLOCKS(N), BLOCK_SIZE>>>(d_positions, d_velocities);

    stateToOrigUnits<<<NUM_BLOCKS(N), BLOCK_SIZE>>>(d_positions, d_velocities, H, DT, N);

    hostTime(memcpy,
             hipMemcpy(state.data(), d_positions, N * sizeof(Vec3), hipMemcpyDeviceToHost));

    hostTime(recordState, stateRecorder.recordPositions(state.begin(), state.begin() + N));

    auto boundsBegin = std::chrono::steady_clock::now();
    if (std::any_of(
            std::execution::par_unseq, state.begin(), state.begin() + N,
            [effectiveBoxSize](const Vec3& pos) { return !isWithinBox(pos, effectiveBoxSize); })) {
      std::cout << "Particle moved outside the grid.\n";
      break;
    }
    auto boundsEnd = std::chrono::steady_clock::now();
    boundsCheckTimeMs += toMs(boundsEnd - boundsBegin);

    stateToCodeUnits<<<NUM_BLOCKS(N), BLOCK_SIZE>>>(d_positions, d_velocities, H, DT, N);

    hostTime(pm,
             pmMethodStep(d_accelerations, d_gridDensity, d_gridDensityFourier, d_gridPotential,
                          d_gridPotentialFourier, d_gridField, d_positions, d_masses, H, DT, G));
    updateVelocities<<<NUM_BLOCKS(N), BLOCK_SIZE>>>(d_velocities, d_accelerations);
  }
  hipDeviceSynchronize();
  stateRecorder.flush();
  std::cout << "reassign density: " << reassignDensityMs << '\n';
  std::cout << "forward FFT: " << forwardFFTMs << '\n';
  std::cout << "find fourier potential: " << findFourierPotentialMs << '\n';
  std::cout << "inverse FFT: " << inverseFFTMs << '\n';
  std::cout << "find field in cells: " << findFieldInCellsMs << '\n';
  std::cout << "update accelerations: " << updateAccelerationsMs << '\n';
  std::cout << "total PM: " << pmTimeMs << '\n';
  std::cout << "memcpy: " << memcpyTimeMs << '\n';
  std::cout << "bounds check: " << boundsCheckTimeMs << '\n';
  std::cout << "record: " << recordStateTimeMs << '\n';

  hipFree(d_masses);
  hipFree(d_positions);
  hipFree(d_velocities);
  hipFree(d_accelerations);

  hipFree(d_gridDensity);
  hipFree(d_gridDensityFourier);
  hipFree(d_gridPotential);
  hipFree(d_gridPotentialFourier);
  hipFree(d_gridField);

  hipEventDestroy(reassignDensityStart);
  hipEventDestroy(reassignDensityStop);

  hipEventDestroy(forwardFFTStart);
  hipEventDestroy(forwardFFTStop);

  hipEventDestroy(findFourierPotentialStart);
  hipEventDestroy(findFourierPotentialStop);

  hipEventDestroy(inverseFFTStart);
  hipEventDestroy(inverseFFTStop);

  hipEventDestroy(scaleAfterInverseStart);
  hipEventDestroy(scaleAfterInverseStop);

  hipEventDestroy(findFieldInCellsStart);
  hipEventDestroy(findFieldInCellsStop);

  hipEventDestroy(updateAccelerationsStart);
  hipEventDestroy(updateAccelerationsStop);
}